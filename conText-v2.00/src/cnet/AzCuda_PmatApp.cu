#include "hip/hip_runtime.h"
/* * * * *
 *  AzCuda_PmatApp.cu
 *  Copyright (C) 2014-2015 Rie Johnson
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program.  If not, see <http://www.gnu.org/licenses/>.
 * * * * */

#include "AzCuda_PmatApp.cuh"

#ifdef __AZ_GPU__
  #include "AzCuda.cuh"
  #include "AzCuda_Pmat.cuh"  /* azc_config */
  static void chk_err(const char *eyec, int bb, int tt) {
    AzCuda::check_error(eyec, bb, tt);   
  }   
#else
  extern bool __doDebug; 
  #include "AzPrint.hpp"
  static bool azc_config(int num, int &bb, int &tt, const char *msg="") { return true; }
  static void chk_err(const char *eyec, int bb, int tt) {
    if (__doDebug) AzPrint::writeln(log_out, eyec); 
  }  
#endif   
  
  /*****  PmatApp  *****/
   /*---  L1L2: sqrt(x^2+d^2)-d  ---*/
  __global__ void azc_l1l2(const AzFloat *src, AzFloat *dst, int num, AzFloat del) {  
    double delsq = del*del; 
    int ix; 
    for (ix = azc_thno; ix < num; ix += azc_thnum) {
      dst[ix] = sqrt(src[ix]*src[ix] + delsq) - del; 
    }
  }

  /*---  L1L2-deriv: x/sqrt(x^2+d^2)  ---*/
  __global__ void azc_add_l1l2deriv(const AzFloat *src, AzFloat *dst, int num, AzFloat del, AzFloat coeff) {  
    double delsq = del*del; 
    int ix; 
    for (ix = azc_thno; ix < num; ix += azc_thnum) {
      dst[ix] += (src[ix]*coeff) / sqrt(src[ix]*src[ix] + delsq); 
    }
  } 
  void azccall_l1l2(const AzFloat *src, AzFloat *dst, int num, AzFloat del) {
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_l1l2"); 
    azc_kernel(azc_l1l2,bb,tt)(src, dst, num, del); 
    chk_err("azccall_l1l2", bb, tt);     
  }  
  void azccall_add_l1l2deriv(const AzFloat *src, AzFloat *dst, int num, AzFloat del, AzFloat coeff) {
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_add_l1l2deriv"); 
    azc_kernel(azc_add_l1l2deriv,bb,tt)(src, dst, num, del, coeff); 
    chk_err("azccall_add_l1l2deriv", bb, tt); 
  } 
  
  /*--------------------------------------------------*/  
  /*---        min(th,max(0,x)) activation         ---*/
  /*--------------------------------------------------*/  
  __global__ void azc_activate_th(int num, AzFloat *elm, AzFloat th) {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) {
      if      (elm[ex] <= 0)           elm[ex] = 0; 
      else if (th > 0 && elm[ex] > th) elm[ex] = th;     
    }
  }                    
  __global__ void azc_activate_th_deriv(int num, AzFloat *elm, AzFloat th,
                             AzFloat *deriv_elm) /* must not be NULL */ {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) {                             
      if (elm[ex] <= 0) {
        elm[ex] = 0; 
        deriv_elm[ex] = 0; 
      } else if (th > 0 && elm[ex] > th) {
        elm[ex] = th; 
        deriv_elm[ex] = 0; 
      } else {
        deriv_elm[ex] = 1; 
      }
    }
  }
  __global__ void azc_activate_rect(int num, AzFloat *elm) {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) if (elm[ex] <= 0) elm[ex] = 0;  
  }                    
  __global__ void azc_activate_rect_deriv(int num, AzFloat *elm,
                             AzFloat *deriv_elm) /* must not be NULL */ {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) {                             
      if (elm[ex] <= 0) {
        elm[ex] = 0; 
        deriv_elm[ex] = 0; 
      } else {
        deriv_elm[ex] = 1; 
      }
    }
  }
  
  /*--------------------------------------------------*/   
  void azccall_activate_th(AzFloat *elm, int num, AzFloat th, 
                           AzFloat *deriv_elm) /* may be NULL */ {
    /* note: single vs multi didn't matter */
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_activate_th"); 
    if (th < 0) {
      if (deriv_elm == NULL) azc_kernel(azc_activate_rect,bb,tt)(num, elm); 
      else                   azc_kernel(azc_activate_rect_deriv,bb,tt)(num, elm, deriv_elm);     
    }
    else {
      if (deriv_elm == NULL) azc_kernel(azc_activate_th,bb,tt)(num, elm, th); 
      else                   azc_kernel(azc_activate_th_deriv,bb,tt)(num, elm, th, deriv_elm); 
    }
    chk_err("azccall_activate_th", bb, tt); 
  }  

  /*--------------------------------------------------*/
  /*---             sigmoid activation             ---*/
  /*--------------------------------------------------*/  
  __global__ void azc_activate_log(AzFloat *elm, int num) {
    int ex; 
    for (ex = azc_thno; ex < num; ex += azc_thnum) {  
      elm[ex] = (AzFloat)1/((AzFloat)1+myexp(-elm[ex])); 
    }
  } 
  /*--------------------------------------------------*/  
  __global__ void azc_activate_log_deriv(AzFloat *elm, int num, 
                                    AzFloat *deriv_elm) {
    int ex; 
    for (ex = azc_thno; ex < num; ex += azc_thnum) {  
      AzFloat ss = (AzFloat)1/((AzFloat)1+myexp(-elm[ex])); 
      elm[ex] = ss; 
      deriv_elm[ex] = ss*(1-ss); 
    }
  } 
  /*--------------------------------------------------*/  
  void azccall_activate_log(AzFloat *elm, int num,
                           AzFloat *deriv_elm) /* may be NULL */ {
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_activate_log"); 
    if (deriv_elm == NULL) azc_kernel(azc_activate_log,bb,tt)(elm, num); 
    else                   azc_kernel(azc_activate_log_deriv,bb,tt)(elm, num, deriv_elm); 
    chk_err("azccall_activate_log", bb, tt); 
  } 
  
  /*------------------------------------------------*/
  /*---             tanh activation              ---*/
  /*------------------------------------------------*/
  __global__ void azc_activate_tanh(AzFloat *elm, int num) {
    int ex; 
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      double e2 = myexp(2*elm[ex]); 
      elm[ex] = (AzFloat)((e2-1)/(e2+1));   
    }
  } 
  /*--------------------------------------------------*/  
  __global__ void azc_activate_tanh_deriv(AzFloat *elm, int num, 
                                  AzFloat *deriv_elm) {
    int ex; 
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      double e2 = myexp(2*elm[ex]); 
      elm[ex] = (AzFloat)((e2-1)/(e2+1));
      deriv_elm[ex] = 4*e2/(e2+1)/(e2+1); 
    }
  }
  /*--------------------------------------------------*/  
  void azccall_activate_tanh(AzFloat *elm, int num,
                             AzFloat *deriv_elm) /* may be NULL */ {
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_activate_tanh"); 
    if (deriv_elm == NULL) azc_kernel(azc_activate_tanh,bb,tt)(elm, num); 
    else                   azc_kernel(azc_activate_tanh_deriv,bb,tt)(elm, num, deriv_elm); 
    chk_err("azccall_activate_tanh", bb, tt); 
  } 
  
  /*--------------------------------------------------*/
  /*---            softplus activation             ---*/
  /*--------------------------------------------------*/  
  __global__ void azc_activate_softplus(AzFloat *elm, int num) {
    int ex; 
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      double e2 = myexp(elm[ex]); 
      elm[ex] = (AzFloat)log(1+e2);   
    }
  } 
  /*--------------------------------------------------*/  
  __global__ void azc_activate_softplus_deriv(AzFloat *elm, int num, 
                                    AzFloat *deriv_elm) {
    int ex; 
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      double e2 = myexp(elm[ex]); 
      elm[ex] = (AzFloat)log(1+e2);  
      deriv_elm[ex] = e2/(e2+1);    
    }
  }
  /*--------------------------------------------------*/  
  void azccall_activate_softplus(AzFloat *elm, int num, 
                                 AzFloat *deriv_elm) /* may be NULL */ {
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_activate_softplus"); 
    if (deriv_elm == NULL) azc_kernel(azc_activate_softplus,bb,tt)(elm, num); 
    else                   azc_kernel(azc_activate_softplus_deriv,bb,tt)(elm, num, deriv_elm); 
    chk_err("azccall_activate_softplus", bb, tt); 
  }   

  /*--------------------------------------------------*/    
  /*------------------------------------------------*/
  __global__ void azc_truncate(AzFloat *elm, int num, AzFloat border) {
    int i; 
    for (i = azc_thno; i < num; i +=azc_thnum) {
      elm[i] = MAX(-border, MIN(border, elm[i])); 
    }
  }
  __global__ void azc_truncate_deriv(AzFloat *elm, int num, AzFloat border, 
                                     AzFloat *deriv_elm) /* must not be NULL */ {
    int i; 
    for (i = azc_thno; i < num; i +=azc_thnum) {
      if (elm[i] < -border) {
        elm[i] = -border; 
        deriv_elm[i] = 0; 
      } 
      else if (elm[i] > border) {
        elm[i] = border; 
        deriv_elm[i] = 0; 
      } 
    }
  }  
  void azccall_truncate(AzFloat *elm, int num, AzFloat border, 
                        AzFloat *deriv_elm) /* may be NULL */ {
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_truncate"); 
    if (deriv_elm == NULL) azc_kernel(azc_truncate,bb,tt)(elm, num, border); 
    else                   azc_kernel(azc_truncate_deriv,bb,tt)(elm, num, border, deriv_elm); 
    chk_err("azccall_truncate", bb, tt); 
  }  

  /*******           For convolutional layers             *******/
  /*------------------------------------------------------------*/
  /*---              filtering/unfiltering                   ---*/
  /*------------------------------------------------------------*/
  __global__ void azc_add_with_map(int num, const azcparam_add_with_map p) 
  {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex;  
      int row = index % p.row_num; 
      index = index / p.row_num; 
      int dx = index / p.width2; 
      int col2 = index % p.width2; 
      if (dx >= p.data_num) continue;  
    
      int base1 = dx*p.width1; 
      int base2 = dx*p.width2; 
  
      const int *col1 = _column(col2, p.a2to1, p.nummax); 
      AzFloat *e2 = _column(base2+col2, p.elm2, p.row_num); 
      int ix; 
      for (ix = 0; ix < p.nummax; ++ix) {
        if (col1[ix] == p.stopper) break; 
        const AzFloat *e1 = _column(base1+col1[ix], p.elm1, p.row_num); 
        e2[row] += e1[row];     
      }
    }
  }

  /*------------------------------------------------------------*/
  void azccall_add_with_map(const azcparam_add_with_map p) {
    int num = p.data_num * p.width2 * p.row_num; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_add_with_map"); 
    azc_kernel(azc_add_with_map,bb,tt)(num, p); 
    chk_err("azccall_add_with_map", bb, tt); 
  }
  
  /*------------------------------------------------------------*/
  /*---                  average pooling                     ---*/
  /*------------------------------------------------------------*/
  __global__ void azc_pooling_avg(int num, const azcparam_pooling_avg p) 
  {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex; 
      int row = index % p.row_num; 
      index = index / p.row_num;   
      int dx = index / p.width2; 
      int col2 = index % p.width2; 
      if (dx >= p.data_num) continue; 

      int base1 = dx*p.width1; 
      int base2 = dx*p.width2; 

      const int *col1 = _column(col2, p.col1_ptr, p.col1_nummax); 
      AzFloat *e2 = _column(base2+col2, p.elm2, p.row_num); 
      int ix; 
      for (ix = 0; ix < p.col1_nummax; ++ix) {
        if (col1[ix] == p.stopper) break; 
        const AzFloat *e1 = _column(base1+col1[ix], p.elm1, p.row_num); 
        e2[row] += e1[row];  
      }
      int col1_num = ix; 
      if (col1_num != 0) e2[row] /= (AzFloat)col1_num; 
    }
  }

  /*------------------------------------------------------------*/
  void azccall_pooling_avg(const azcparam_pooling_avg p) {
    int num = p.data_num * p.width2 * p.row_num; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_pooling_avg"); 
    azc_kernel(azc_pooling_avg,bb,tt)(num, p); 
    chk_err("azccall_pooling_avg", bb, tt);         
  }              
              
  /*------------------------------------------------------------*/
  /*---                  average unpooling                   ---*/
  /*------------------------------------------------------------*/
  __global__ void azc_unpooling_avg(int num, const azcparam_unpooling_avg p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex;     
      int row = index % p.row_num; 
      index = index / p.row_num;         
      int dx = index / p.width1; 
      int col1 = index % p.width1; 
      if (dx >= p.data_num) continue; 

      int base1 = dx*p.width1; 
      int base2 = dx*p.width2;   

      const int *col2 = _column(col1, p.col2_ptr, p.col2_nummax); 
      int ix; 
      for (ix = 0; ix < p.col2_nummax; ++ix) {
        if (col2[ix] == p.stopper) break;       
        AzFloat *e1 = _column(base1+col1, p.elm1, p.row_num); 
        const AzFloat *e2 = _column(base2+col2[ix], p.elm2, p.row_num); 
        AzFloat denomi = (AzFloat)p.col2_to_num[col2[ix]]; 
        if (denomi != 0) e1[row] += e2[row]/denomi; 
      }
    }
  }

  /*------------------------------------------------------------*/
  void azccall_unpooling_avg(const azcparam_unpooling_avg p) {
    int num = p.data_num * p.width1 * p.row_num; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_unpooling_avg"); 
    azc_kernel(azc_unpooling_avg,bb,tt)(num, p); 
    chk_err("azccall_unpooling_avg", bb, tt);         
  }

  /*------------------------------------------------------------*/
  /*---                     l2 pooling                       ---*/
  /*------------------------------------------------------------*/
  __global__ void azc_pooling_l2(int num, const azcparam_pooling_l2 p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex;   
      int row = index % p.row_num; 
      index = index / p.row_num; 
      int dx = index / p.width2; 
      int col2 = index % p.width2; 
      if (dx >= p.data_num) continue; 

      int base1 = dx*p.width1; 
      int base2 = dx*p.width2; 

      const int *col1 = _column(col2, p.col1_ptr, p.col1_nummax); 
      AzFloat *e2 = _column(base2+col2, p.elm2, p.row_num); 
      int ix; 
      for (ix = 0; ix < p.col1_nummax; ++ix) {
        if (col1[ix] == p.stopper) break; 
        const AzFloat *e1 = _column(base1+col1[ix], p.elm1, p.row_num); 
        e2[row] += e1[row]*e1[row]; 
      }
      e2[row] = sqrt(e2[row]); 
    }
  }

  /*------------------------------------------------------------*/
  void azccall_pooling_l2(const azcparam_pooling_l2 p) {
    int num = p.data_num * p.width2 * p.row_num; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_pooling_l2");  
    azc_kernel(azc_pooling_l2,bb,tt)(num, p); 
    chk_err("azccall_pooling_l2", bb, tt);         
  }
 
  /*------------------------------------------------------------*/
  /*---                    l2 unpooling                      ---*/
  /*------------------------------------------------------------*/
  __global__ void azc_unpooling_l2(int num, const azcparam_unpooling_l2 p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex; 
      int row = index % p.row_num; 
      index = index / p.row_num;    
      int dx = index / p.width1; 
      int col1 = index % p.width1; 
      if (dx >= p.data_num) continue; 

      int base1 = dx*p.width1; 
      int base2 = dx*p.width2;   

      AzFloat *e1 = _column(base1+col1, p.elm1, p.row_num); 
      const AzFloat *org_e1 = _column(base1+col1, p.org_elm1, p.row_num); 
      const int *col2 = _column(col1, p.col2_ptr, p.col2_nummax); 
      int ix; 
      for (ix = 0; ix < p.col2_nummax; ++ix) {
        if (col2[ix] == p.stopper) break;       
        const AzFloat *e2 = _column(base2+col2[ix], p.elm2, p.row_num); 
        const AzFloat *org_e2 = _column(base2+col2[ix], p.org_elm2, p.row_num); 
        if (org_e2[row] != 0) e1[row] += (e2[row] * org_e1[row] / org_e2[row]); 
      }
    }
  }

  /*------------------------------------------------------------*/
  void azccall_unpooling_l2(const azcparam_unpooling_l2 p) {
    int num = p.data_num * p.width1 * p.row_num; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_unpooling_l2"); 
    azc_kernel(azc_unpooling_l2,bb,tt)(num, p); 
    chk_err("azccall_unpooling_l2", bb, tt);         
  }
  
  /*------------------------------------------------------------*/
  /*---                    max pooling                       ---*/
  /*------------------------------------------------------------*/
  __global__ void azc_pooling_max(int num, const azcparam_pooling_max p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex; 
      int row = index % p.row_num; 
      index = index / p.row_num;         
      int dx = index / p.width2; 
      int col2 = index % p.width2; 
      if (dx >= p.data_num) continue; 
  
      int base1 = dx*p.width1; 
      int base2 = dx*p.width2; 

      const int *col1 = _column(col2, p.col1_ptr, p.col1_nummax); 
      int *chosen = (p.chosen_ptr != NULL) ? _column(col2+base2, p.chosen_ptr, p.row_num) : NULL; 
      AzFloat *max_elm = _column(col2+base2, p.elm2, p.row_num); 
      int ix; 
      for (ix = 0; ix < p.col1_nummax; ++ix) {
        if (col1[ix] == p.stopper) break; 
        const AzFloat *e1 = _column(col1[ix]+base1, p.elm1, p.row_num); 
        if (ix == 0 || e1[row] > max_elm[row]) {
          max_elm[row] = e1[row]; 
          if (chosen != NULL) chosen[row] = col1[ix]; 
        }
      }    
    }                         
  }

  /*------------------------------------------------------------*/
  void azccall_pooling_max(const azcparam_pooling_max p) {
    int num = p.data_num * p.width2 * p.row_num; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_pooling_max"); 
    azc_kernel(azc_pooling_max,bb,tt)(num, p); 
    chk_err("azccall_pooling_max", bb, tt);         
  }

  /*------------------------------------------------------------*/  
  /*---                   max unpooling                      ---*/
  /*------------------------------------------------------------*/
  /* Note: assume overlapping pooling */
  /* thread: portions of rows of one data point  */
  __global__ void azc_unpooling_max(int num, const azcparam_unpooling_max p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex;   
      int row = index % p.row_num; 
      index = index / p.row_num;    
      int dx = index % p.data_num; 
      if (dx >= p.data_num) continue; 
  
      int base1 = dx*p.width1; 
      int base2 = dx*p.width2; 
    
      int col2; 
      for (col2 = 0; col2 < p.width2; ++col2) {
        const AzFloat *e2 = _column(col2+base2, p.elm2, p.row_num); 
        const int *chosen = _column(col2+base2, p.ptr_chosen, p.row_num); 
        int col1 = chosen[row]; 
        if (col1 >= 0) { /* 3/11/2014: for variable-length pooling */
          (_column(col1+base1, p.elm1, p.row_num))[row] += e2[row]; 
        }
      }
    }
  }
  
  /*------------------------------------------------------------*/
  void azccall_unpooling_max(const azcparam_unpooling_max p) {
    int num = p.data_num * p.row_num; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_unpooling_max"); 
    azc_kernel(azc_unpooling_max,bb,tt)(num, p); 
    chk_err("azccall_unpooling_max", bb, tt);         
  }

  /*------------------------------------------------------------*/    
  /*---      rearrange (for locally-connected weights)       ---*/
  /*------------------------------------------------------------*/    
  __global__ void azc_rearrange(int num, const azcparam_rearrange p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex;   
      int row = index % p.rnum; 
      index = index / p.rnum;         
      int dx = index / p.loc_num; 
      int loc = index % p.loc_num; 
      if (dx >= p.d_num) continue; 

      const AzFloat *e1 = _column(p.loc_num*dx + loc, p.elm1, p.rnum); 
      AzFloat *e2 = _column(p.d_num*loc + dx, p.elm2, p.rnum); 
      e2[row] = e1[row];    
    }
  }

  /*------------------------------------------------------------*/
  void azccall_rearrange(const azcparam_rearrange p) {
    int num = p.d_num * p.loc_num * p.rnum; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_rearrange"); 
    azc_kernel(azc_rearrange,bb,tt)(num, p); 
    chk_err("azccall_rearrange", bb, tt);         
  }    

  /*------------------------------------------------------------*/  
  /*------------------------------------------------------------*/      
  __global__ void azc_undo_rearrange(int num, const azcparam_undo_rearrange p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex;   
      int row = index % p.rnum; 
      index = index / p.rnum;         
      int dx = index / p.loc_num; 
      int loc = index % p.loc_num; 
      if (dx >= p.d_num) continue; 

      AzFloat *e1 = _column(p.loc_num*dx + loc, p.elm1, p.rnum); 
      const AzFloat *e2 = _column(p.d_num*loc + dx, p.elm2, p.rnum);     
      e1[row] = e2[row]; 
    }
  }

  /*------------------------------------------------------------*/
  void azccall_undo_rearrange(const azcparam_undo_rearrange p) {
    int num = p.d_num * p.loc_num * p.rnum; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_undo_rearrange"); 
    azc_kernel(azc_undo_rearrange,bb,tt)(num, p); 
    chk_err("azccall_undo_rearrange", bb, tt);         
  }   

  /*------------------------------------------------------------*/
  /*-- local response normalization across neurons  (cmrnorm) --*/
  /*------------------------------------------------------------*/
  __global__ void azc_resnorm_crossmap(int num, const azcparam_resnorm_crossmap p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex;   
      int row = index % p.rnum; 
      int col = index / p.rnum; 
      if (col >= p.cnum) continue; 
  
      int sz, rr0, rr1; 
      if (p.size < p.rnum) {
        int halfsize = p.size / 2; 
        sz = halfsize*2 + 1; 
        sz = 1; /* to match with cuda-convnet */
        rr0 = row - halfsize;
        rr1 = row + halfsize; 
      }
      else {
        sz = 1; 
        rr0 = 0; 
        rr1 = p.rnum-1; 
      }
    
      int base = col*p.rnum; 
      const AzFloat *input = p.elm + base; 
      AzFloat *oneplussqavg = p.elm_oneplussqavg + base; 
      AzFloat *normalized = p.elm_normalized + base;   

      AzFloat sqsum = 0; 
      int rr; 
      for (rr = rr0; rr <= rr1; ++rr) {
        int myrr = (rr+p.rnum) % p.rnum; 
        sqsum += input[myrr]*input[myrr]; 
      }
      oneplussqavg[row] = p.one+p.alpha*sqsum/(double)sz; 
      normalized[row] *= pow(oneplussqavg[row], -p.beta); 
    }
  }
  
  /*------------------------------------------------------------*/
  void azccall_resnorm_crossmap(const azcparam_resnorm_crossmap p) {
    int num = p.cnum * p.rnum; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_resnorm_crossmap");  
    azc_kernel(azc_resnorm_crossmap,bb,tt)(num, p); 
    chk_err("azccall_resnorm_crossmap", bb, tt);         
  }
  
  /*------------------------------------------------------------*/
  /*-- local response normalization across channels (cmrnorm) --*/
  /*------------------------------------------------------------*/
  __global__ void azc_resnorm_crossmap_all(int num, const azcparam_resnorm_crossmap p, 
                  const AzFloat *col_sqsum) {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex;   
      int row = index % p.rnum; 
      int col = index / p.rnum; 
      if (col >= p.cnum) continue; 

      int base = col*p.rnum; 
      AzFloat *oneplussqavg = p.elm_oneplussqavg + base; 
      AzFloat *normalized = p.elm_normalized + base;   
      
      oneplussqavg[row] = p.one+p.alpha*col_sqsum[col]; 
      normalized[row] *= pow(oneplussqavg[row], -p.beta); 
    }
  }
  
  /*------------------------------------------------------------*/
  void azccall_resnorm_crossmap_all(const azcparam_resnorm_crossmap p, const AzFloat *col_sqsum) {
    int num = p.cnum * p.rnum; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_resnorm_crossmap_all");  
    azc_kernel(azc_resnorm_crossmap_all,bb,tt)(num, p, col_sqsum); 
    chk_err("azccall_resnorm_crossmap_all", bb, tt);         
  }  
  
  /*-------------------------------------------------------------*/  
  /*---  prep for undoing response normalization (cross map)  ---*/
  /*-------------------------------------------------------------*/
  /* tmp <- g_k * (-2 alpha beta)/N_k * v_k(1 + alpha/N sum_i v_i^2)^{-beta-1} */
  /*     =  (-2 alpha beta f_k g_k)/(N_k d_k) */
  /*  d_k := 1 + alpha/N_k sum_i v_i^2 */
  /*  f_k := v_k d_k^{-beta} */
  __global__ void azc_prep_unresnorm_crossmap(int num, const azcparam_prep_unresnorm_crossmap p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex;   
      int row = index % p.rnum; 
      int col = index / p.rnum; 
      if (col >= p.cnum) continue; 

      int sz = 1; 
      int base = p.rnum*col; 
      const AzFloat *grad = p.elm_grad + base; 
      const AzFloat *aft = p.elm_aft + base; 
      const AzFloat *oneplussqavg = p.elm_oneplussqavg + base; 
      AzFloat *tmp = p.elm_tmp + base; 

      tmp[row] = (-2*p.alpha*p.beta * aft[row]*grad[row]) / (oneplussqavg[row] * (double)sz); 
    }
  }

  /*------------------------------------------------------------*/
  void azccall_prep_unresnorm_crossmap(const azcparam_prep_unresnorm_crossmap p) {
    int num = p.cnum * p.rnum; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_prep_unresnorm_crossmap"); 
    azc_kernel(azc_prep_unresnorm_crossmap,bb,tt)(num, p); 
    chk_err("azccall_prep_unresnorm_crossmap", bb, tt);         
  }  

  /*-------------------------------------------------------------*/  
  /*---      undo response normalization (cross map)        ---*/
  /*-------------------------------------------------------------*/
  /* v_j sum_k (-2 alpha beta f_k g_k)/(N_k d_k)  +  (f_j g_j) / v_j */
  /*------------------------------------------------------------*/
  __global__ void azc_unresnorm_crossmap(int num, const azcparam_unresnorm_crossmap p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex; 
      int row = index % p.rnum; 
      int col = index / p.rnum; 
      if (col >= p.cnum) continue; 

      int rr0, rr1; 
      if (p.size < p.rnum) {
        int halfsize = p.size / 2; 
        rr0 = row - halfsize;
        rr1 = row + halfsize; 
      }
      else {
        rr0 = 0; 
        rr1 = p.rnum-1; 
      }
    
      int base = col*p.rnum; 
      const AzFloat *tmp = p.elm_tmp + base;  
      const AzFloat *bef = p.elm_bef + base;  
      const AzFloat *oneplussqavg = p.elm_oneplussqavg + base;       
      const AzFloat *grad = p.elm_grad + base;  
      AzFloat *out = p.elm_out + base; 

      AzFloat val = 0; 
      if (bef[row] != 0) {
        int rr; 
        for (rr = rr0; rr <= rr1; ++rr) { /* neighbor relationship is mutual */
          int myrr = (rr+p.rnum)%p.rnum; 
          val += tmp[myrr]; 
        }
      }
      out[row] = bef[row]*val + grad[row]*pow(oneplussqavg[row],-p.beta); 
    }
  }

  /*------------------------------------------------------------*/
  void azccall_unresnorm_crossmap(const azcparam_unresnorm_crossmap p) {
    int num = p.cnum * p.rnum; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_unresnorm_crossmap");  
    azc_kernel(azc_unresnorm_crossmap,bb,tt)(num, p); 
    chk_err("azccall_unresnorm_crossmap", bb, tt);         
  } 
  
  /*------------------------------------------------------------*/
  __global__ void azc_unresnorm_crossmap_all(int num, const azcparam_unresnorm_crossmap p, const AzFloat *tmp_colSum) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex; 
      int row = index % p.rnum; 
      int col = index / p.rnum; 
      if (col >= p.cnum) continue; 

      int base = col*p.rnum; 
      const AzFloat *bef = p.elm_bef + base;  
      const AzFloat *oneplussqavg = p.elm_oneplussqavg + base;       
      const AzFloat *grad = p.elm_grad + base;  
      AzFloat *out = p.elm_out + base; 

      out[row] = bef[row]*tmp_colSum[col] + grad[row]*pow(oneplussqavg[row],-p.beta); 
    }
  }

  /*------------------------------------------------------------*/
  void azccall_unresnorm_crossmap_all(const azcparam_unresnorm_crossmap p, const AzFloat *tmp_colSum) {
    int num = p.cnum * p.rnum; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_unresnorm_crossmap_all");  
    azc_kernel(azc_unresnorm_crossmap_all,bb,tt)(num, p, tmp_colSum); 
    chk_err("azccall_unresnorm_crossmap_all", bb, tt);         
  } 
  
  /*------------------------------------------------------------*/
  /*---            response normalization (local)            ---*/
  /*------------------------------------------------------------*/  
  __global__ void azc_resnorm_local(int num, const azcparam_resnorm_local p) {
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex;  
      int row = index % p.rnum; 
      index = index / p.rnum;    
      int dx = index / p.cnum; 
      int col = index % p.cnum; 
      if (dx >= p.data_num) continue;  

      int base = dx*p.cnum; 
    
      /*---  compute  1 + alpha avg_i v_i^2  ---*/
      const int *neigh = _column(col, p.neighbors, p.nummax); 
      AzFloat *oneplussqavg = _column(base+col, p.elm_oneplussqavg, p.rnum); 
      AzFloat *normalized = _column(base+col, p.elm_normalized, p.rnum); 
      int ix; 
      for (ix = 0; ix < p.nummax; ++ix) {
        if (neigh[ix] == p.stopper) break; 
        const AzFloat *neigh_val = _column(base+neigh[ix], p.elm, p.rnum); 
        oneplussqavg[row] += neigh_val[row]*neigh_val[row]; 
      }
      int sz = p.neigh_sz[col]; 
  
      sz = 1; /* to match with cuda-convnet */
  
      AzFloat coeff = 1; 
      if (sz != 0) {
        coeff = p.alpha / (AzFloat)sz; 
      }
      oneplussqavg[row] *= coeff; 
      oneplussqavg[row] += 1; 
      /*---  multiply (1 + alpha avg_i v_i^2)^{-beta}  ---*/
      normalized[row] *= pow(oneplussqavg[row], -p.beta); 
    }
  }

  /*------------------------------------------------------------*/
  void azccall_resnorm_local(const azcparam_resnorm_local p) {
    int num = p.data_num * p.cnum * p.rnum; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_resnorm_local"); 
    azc_kernel(azc_resnorm_local,bb,tt)(num, p); 
    chk_err("azccall_resnorm_local", bb, tt);         
  } 

  /*------------------------------------------------------------*/  
  /*---   prep for undoing response normalization (local)    ---*/
  /*------------------------------------------------------------*/
  /* tmp <- g_k * (-2 alpha beta)/N_k * v_k(1 + alpha/N sum_i v_i^2)^{-beta-1} */
  /*     =  (-2 alpha beta f_k g_k)/(N_k d_k)                   */
  /*  d_k := 1 + alpha/N_k sum_i v_i^2                          */
  /*  f_k := v_k d_k^{-beta}                                    */
  /*------------------------------------------------------------*/
  __global__ void azc_prep_unresnorm_local(int num, const azcparam_prep_unresnorm_local p) { 
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex; 
      int row = index % p.rnum; 
      index = index / p.rnum;    
      int dx = index / p.cnum; 
      int col = index % p.cnum; 
      if (dx >= p.data_num) continue; 

      int base0 = dx*p.rnum*p.cnum; 
      int base = base0 + p.rnum*col; 
      const AzFloat *grad = p.elm_grad + base; 
      const AzFloat *aft = p.elm_aft + base; 
      const AzFloat *oneplussqavg = p.elm_oneplussqavg + base; 
      AzFloat *tmp = p.elm_tmp + base; 
      int sz = p.neigh_sz[col]; 

      sz = 1; /* to match with cuda-convnet */

      tmp[row] = (-2*p.alpha*p.beta * aft[row]*grad[row]) / (oneplussqavg[row] * (AzFloat)sz); 
    }
  }

  /*------------------------------------------------------------*/
  void azccall_prep_unresnorm_local(const azcparam_prep_unresnorm_local p) {
    int num = p.data_num * p.cnum * p.rnum; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_prep_unresnorm_local"); 
    azc_kernel(azc_prep_unresnorm_local,bb,tt)(num, p);   
    chk_err("azccall_prep_unresnorm_local", bb, tt);         
  }    

  /*------------------------------------------------------------*/  
  /*---          undo response normalization (local)         ---*/  
  /* v_j sum_k (-2 alpha beta f_k g_k)/(N_k d_k)  +  (f_j g_j) / v_j */
  /*------------------------------------------------------------*/
  __global__ void azc_unresnorm_local(int num, const azcparam_unresnorm_local p) { 
    int ex;   
    for (ex = azc_thno; ex < num; ex += azc_thnum) {
      int index = ex; 
      int row = index % p.rnum; 
      index = index / p.rnum;    
      int dx = index / p.cnum; 
      int col = index % p.cnum; 
      if (dx >= p.data_num) continue; 

      int base0 = dx*p.rnum*p.cnum; 
      const AzFloat *tmp = p.elm_tmp + base0;  

      int base = base0 + col*p.rnum; 
      AzFloat *out = p.elm_out + base; 
      const AzFloat *bef = p.elm_bef + base;  
      const AzFloat *oneplussqavg = p.elm_oneplussqavg + base;       
      const AzFloat *grad = p.elm_grad + base;
    
      const int *whose_neigh = _column(col, p.whose_neighbor, p.nummax); 

      AzFloat val = 0; 
      if (bef[row] != 0) {
        int ix; 
        for (ix = 0; ix < p.nummax; ++ix) {    
          int kx = whose_neigh[ix]; 
          if (kx == p.stopper) break;         
          val += _entry(row, kx, tmp, p.rnum); 
        }       
      }
      out[row] = bef[row]*val + grad[row]*pow(oneplussqavg[row], -p.beta);  
    }
  }

  /*------------------------------------------------------------*/
  void azccall_unresnorm_local(const azcparam_unresnorm_local p) {
    int num = p.data_num * p.cnum * p.rnum; 
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_unresnorm_local"); 
    azc_kernel(azc_unresnorm_local,bb,tt)(num, p); 
    chk_err("azccall_unresnorm_local", bb, tt);         
  } 

  /*------------------------------------------------*/
  /* loss: f(p)=log(1+exp(-zp)) for y in {0,1}      */
  /* derivative: f'(p)=-z/(exp(zp)+1)               */
  /* z=2y-1 for y in {0,1}; z=y for y in {-1,1}     */                          
  /*------------------------------------------------*/
  __global__ void azc_binlogi_deriv(bool is_01, 
             const AzFloat *p, const AzFloat *y, int num, 
             AzFloat *ld, AzFloat *loss) {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) {    
      AzFloat pp = p[ex]; 
      AzFloat yy = (is_01) ? 2*y[ex]-1 : y[ex]; 
      AzFloat ee = exp(yy*pp); 
      ld[ex] = -yy/(ee+1); /* -z/(exp(zp)+1) */
      if (loss != NULL) {
        loss[ex] = log(1+1/ee);  /* log(1+exp(-zp)) */
      }
    }
  }
  /*------------------------------------------------------------*/
  void azccall_binlogi_deriv(bool is_01, 
             const AzFloat *p, const AzFloat *y, int num, 
             AzFloat *ld, AzFloat *loss) {
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_binlogi_deriv"); 
    azc_kernel(azc_binlogi_deriv,bb,tt)(is_01, p, y, num, ld, loss); 
    chk_err("azccall_binlogi_deriv", bb, tt);         
  } 
  /*------------------------------------------------------------*/  
  __global__ void azc_binlogi_loss(bool is_01, 
             const AzFloat *p, const AzFloat *y, int num, 
             AzFloat *loss) {
    for (int ex = azc_thno; ex < num; ex += azc_thnum) {    
      AzFloat pp = p[ex]; 
      AzFloat yy = (is_01) ? 2*y[ex]-1 : y[ex]; 
      loss[ex] = log(1+exp(-yy*pp));  /* log(1+exp(-zp)) */
    }
  }  
  /*------------------------------------------------------------*/
  void azccall_binlogi_loss(bool is_01, 
             const AzFloat *p, const AzFloat *y, int num, 
             AzFloat *loss) {
    int bb, tt; 
    azc_config(num, bb, tt, "azccall_binlogi_loss"); 
    azc_kernel(azc_binlogi_loss,bb,tt)(is_01, p, y, num, loss); 
    chk_err("azccall_binlogi_loss", bb, tt);         
  }  
